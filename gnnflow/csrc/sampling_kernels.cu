#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include "common.h"
#include "sampling_kernels.h"
#include "utils.h"

namespace gnnflow {

__global__ void SampleLayerRecentKernel(
    const DoublyLinkedList* node_table, std::size_t num_nodes, bool prop_time,
    const NIDType* root_nodes, const TimestampType* root_timestamps,
    uint32_t snapshot_idx, uint32_t num_snapshots,
    TimestampType snapshot_time_window, uint32_t num_root_nodes,
    uint32_t fanout, NIDType* src_nodes, EIDType* eids,
    TimestampType* timestamps, TimestampType* delta_timestamps,
    uint32_t* num_sampled) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_root_nodes) {
    return;
  }

  NIDType nid = root_nodes[tid];
  TimestampType root_timestamp = root_timestamps[tid];
  TimestampType start_timestamp, end_timestamp;
  if (num_snapshots == 1) {
    if (abs(snapshot_time_window) < 1e-6) {
      start_timestamp = 0;
    } else {
      start_timestamp = root_timestamp - snapshot_time_window;
    }
    end_timestamp = root_timestamp;
  } else {
    end_timestamp = root_timestamp -
                    (num_snapshots - snapshot_idx - 1) * snapshot_time_window;
    start_timestamp = end_timestamp - snapshot_time_window;
  }

  // NB: the tail block is the newest block
  auto curr = node_table[nid].tail;
  uint32_t offset = tid * fanout;
  int start_idx, end_idx;
  uint32_t sampled = 0;
  while (curr != nullptr && curr->capacity > 0 && sampled < fanout) {
    if (end_timestamp < curr->start_timestamp) {
      // search in the previous block
      curr = curr->prev;
      continue;
    }

    if (start_timestamp > curr->end_timestamp) {
      // no need to search in the previous block
      break;
    }

    // search in the current block
    if (start_timestamp >= curr->start_timestamp &&
        end_timestamp <= curr->end_timestamp) {
      // all edges in the current block
      LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
      LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
    } else if (start_timestamp < curr->start_timestamp &&
               end_timestamp <= curr->end_timestamp) {
      // only the edges before end_timestamp are in the current block
      start_idx = 0;
      LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
    } else if (start_timestamp > curr->start_timestamp &&
               end_timestamp > curr->end_timestamp) {
      // only the edges after start_timestamp are in the current block
      LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
      end_idx = curr->size;
    } else {
      // the whole block is in the range
      start_idx = 0;
      end_idx = curr->size;
    }

    // copy the edges to the output
    for (int i = end_idx - 1; sampled < fanout && i >= start_idx; --i) {
      src_nodes[offset + sampled] = curr->dst_nodes[i];
      eids[offset + sampled] = curr->eids[i];
      auto ts = curr->timestamps[i];
      timestamps[offset + sampled] = ts;
      delta_timestamps[offset + sampled] = root_timestamp - ts;
      ++sampled;
    }

    curr = curr->prev;
  }

  num_sampled[tid] = sampled;

  while (sampled < fanout) {
    src_nodes[offset + sampled] = kInvalidNID;
    ++sampled;
  }
}

__global__ void SampleLayerUniformKernel(
    const DoublyLinkedList* node_table, std::size_t num_nodes, bool prop_time,
    hiprandState_t* rand_states, uint64_t seed, uint32_t offset_per_thread,
    const NIDType* root_nodes, const TimestampType* root_timestamps,
    uint32_t snapshot_idx, uint32_t num_snapshots,
    TimestampType snapshot_time_window, uint32_t num_root_nodes,
    uint32_t fanout, NIDType* src_nodes, EIDType* eids,
    TimestampType* timestamps, TimestampType* delta_timestamps,
    uint32_t* num_sampled) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_root_nodes) {
    return;
  }

  extern __shared__ SamplingRange ranges[];

  NIDType nid = root_nodes[tid];
  TimestampType root_timestamp = root_timestamps[tid];
  TimestampType start_timestamp, end_timestamp;
  if (num_snapshots == 1) {
    if (abs(snapshot_time_window) < 1e-6) {
      start_timestamp = 0;
    } else {
      start_timestamp = root_timestamp - snapshot_time_window;
    }
    end_timestamp = root_timestamp;
  } else {
    end_timestamp = root_timestamp -
                    (num_snapshots - snapshot_idx - 1) * snapshot_time_window;
    start_timestamp = end_timestamp - snapshot_time_window;
  }

  auto& list = node_table[nid];
  uint32_t num_candidates = 0;

  // NB: the tail block is the newest block
  auto curr = list.tail;
  int start_idx, end_idx;
  int curr_idx = 0;
  const int offset_by_thread = offset_per_thread * threadIdx.x;
  while (curr != nullptr && curr->capacity > 0) {
    if (end_timestamp < curr->start_timestamp) {
      // search in the prev block
      curr = curr->prev;
      curr_idx += 1;
      continue;
    }

    if (start_timestamp > curr->end_timestamp) {
      // no need to search in the prev block
      break;
    }

    // search in the current block
    if (start_timestamp >= curr->start_timestamp &&
        end_timestamp <= curr->end_timestamp) {
      // all edges in the current block
      LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
      LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
    } else if (start_timestamp < curr->start_timestamp &&
               end_timestamp <= curr->end_timestamp) {
      // only the edges before end_timestamp are in the current block
      start_idx = 0;
      LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
    } else if (start_timestamp > curr->start_timestamp &&
               end_timestamp > curr->end_timestamp) {
      // only the edges after start_timestamp are in the current block
      LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
      end_idx = curr->size;
    } else {
      // the whole block is in the range
      start_idx = 0;
      end_idx = curr->size;
    }

    if (curr_idx < offset_per_thread) {
      ranges[offset_by_thread + curr_idx].start_idx = start_idx;
      ranges[offset_by_thread + curr_idx].end_idx = end_idx;
    }

    num_candidates += end_idx - start_idx;
    curr = curr->prev;
    curr_idx += 1;
  }

  uint32_t indices[kMaxFanout];
  uint32_t to_sample = min(fanout, num_candidates);
  for (uint32_t i = 0; i < to_sample; i++) {
    indices[i] = hiprand(rand_states + tid) % num_candidates;
  }
  BubbleSort(indices, to_sample);

  uint32_t sampled = 0;
  uint32_t offset = tid * fanout;

  curr = list.tail;
  curr_idx = 0;
  uint32_t cumsum = 0;
  while (curr != nullptr && curr->capacity > 0) {
    if (end_timestamp < curr->start_timestamp) {
      // search in the prev block
      curr = curr->prev;
      curr_idx += 1;
      continue;
    }

    if (start_timestamp > curr->end_timestamp) {
      // no need to search in the prev block
      break;
    }

    if (curr_idx < offset_per_thread) {
      start_idx = ranges[offset_by_thread + curr_idx].start_idx;
      end_idx = ranges[offset_by_thread + curr_idx].end_idx;
    } else {
      // search in the current block
      if (start_timestamp >= curr->start_timestamp &&
          end_timestamp <= curr->end_timestamp) {
        // all edges in the current block
        LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
        LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
      } else if (start_timestamp < curr->start_timestamp &&
                 end_timestamp <= curr->end_timestamp) {
        // only the edges before end_timestamp are in the current block
        start_idx = 0;
        LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
      } else if (start_timestamp > curr->start_timestamp &&
                 end_timestamp > curr->end_timestamp) {
        // only the edges after start_timestamp are in the current block
        LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
        end_idx = curr->size;
      } else {
        // the whole block is in the range
        start_idx = 0;
        end_idx = curr->size;
      }
    }

    auto idx = indices[sampled] - cumsum;
    while (sampled < to_sample && idx < end_idx - start_idx) {
      // start from end_idx (newer edges)
      src_nodes[offset + sampled] = curr->dst_nodes[end_idx - idx - 1];
      eids[offset + sampled] = curr->eids[end_idx - idx - 1];
      auto ts = curr->timestamps[end_idx - idx - 1];
      timestamps[offset + sampled] =
          prop_time ? root_timestamp : ts;
      delta_timestamps[offset + sampled] = root_timestamp - ts;
      idx = indices[++sampled] - cumsum;
    }

    if (sampled >= to_sample) {
      break;
    }

    cumsum += end_idx - start_idx;
    curr = curr->prev;
    curr_idx += 1;
  }

  num_sampled[tid] = sampled;

  while (sampled < fanout) {
    src_nodes[offset + sampled] = kInvalidNID;
    ++sampled;
  }
}

}  // namespace gnnflow
