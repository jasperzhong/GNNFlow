#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include <algorithm>
#include <cmath>
#include <cstdint>
#include <numeric>
#include <rmm/device_vector.hpp>

#include "common.h"
#include "sampling_kernels.h"
#include "temporal_sampler.h"
#include "utils.h"

namespace dgnn {

TemporalSampler::TemporalSampler(const DynamicGraph& graph,
                                 const std::vector<uint32_t>& fanouts,
                                 SamplingPolicy sampling_policy,
                                 uint32_t num_snapshots,
                                 float snapshot_time_window, bool prop_time,
                                 uint64_t seed)
    : graph_(graph),
      fanouts_(fanouts),
      sampling_policy_(sampling_policy),
      num_snapshots_(num_snapshots),
      snapshot_time_window_(snapshot_time_window),
      prop_time_(prop_time),
      num_layers_(fanouts.size()),
      seed_(seed) {
  if (num_snapshots_ == 1 && std::fabs(snapshot_time_window_) > 0.0f) {
    LOG(WARNING) << "Snapshot time window must be 0 when num_snapshots = 1. "
                    "Ignore the snapshot time window.";
  }
}

std::vector<SamplingResult> TemporalSampler::RootInputToSamplingResult(
    const std::vector<NIDType>& dst_nodes,
    const std::vector<TimestampType>& dst_timestamps) {
  std::vector<SamplingResult> sampling_results(num_snapshots_);
  for (int snapshot = 0; snapshot < num_snapshots_; ++snapshot) {
    auto& sampling_result = sampling_results[snapshot];
    sampling_result.all_nodes.insert(sampling_result.all_nodes.end(),
                                     dst_nodes.begin(), dst_nodes.end());
    sampling_result.all_timestamps.insert(sampling_result.all_timestamps.end(),
                                          dst_timestamps.begin(),
                                          dst_timestamps.end());
  }
  return sampling_results;
}

std::vector<SamplingResult> TemporalSampler::SampleLayer(
    uint32_t layer, const std::vector<SamplingResult>& prev_sampling_results) {
  CHECK_EQ(prev_sampling_results.size(), num_snapshots_);
  // host input
  std::vector<NIDType> root_nodes;
  std::vector<TimestampType> root_timestamps;
  std::vector<uint32_t> cumsum_num_nodes;
  uint32_t cumsum = 0;

  // from old to new
  for (int snapshot = 0; snapshot < num_snapshots_; ++snapshot) {
    root_nodes.insert(root_nodes.end(),
                      prev_sampling_results.at(snapshot).all_nodes.begin(),
                      prev_sampling_results.at(snapshot).all_nodes.end());

    root_timestamps.insert(
        root_timestamps.end(),
        prev_sampling_results.at(snapshot).all_timestamps.begin(),
        prev_sampling_results.at(snapshot).all_timestamps.end());

    uint32_t num_nodes = prev_sampling_results.at(snapshot).all_nodes.size();
    cumsum += num_nodes;
    cumsum_num_nodes.push_back(cumsum);
  }

  std::size_t total_input_size =
      root_nodes.size() * sizeof(NIDType) +
      root_timestamps.size() * sizeof(TimestampType) +
      cumsum_num_nodes.size() * sizeof(uint32_t);

  char* tmp_host_buffer = new char[total_input_size];
  std::copy(root_nodes.begin(), root_nodes.end(),
            reinterpret_cast<NIDType*>(tmp_host_buffer));
  std::copy(root_timestamps.begin(), root_timestamps.end(),
            reinterpret_cast<TimestampType*>(
                tmp_host_buffer + root_nodes.size() * sizeof(NIDType)));
  std::copy(cumsum_num_nodes.begin(), cumsum_num_nodes.end(),
            reinterpret_cast<uint32_t*>(
                tmp_host_buffer + root_nodes.size() * sizeof(NIDType) +
                root_timestamps.size() * sizeof(TimestampType)));

  // device input
  auto mr = rmm::mr::get_current_device_resource();
  char* d_input = reinterpret_cast<char*>(mr->allocate(total_input_size));
  CUDA_CALL(hipMemcpy(d_input, tmp_host_buffer, total_input_size,
                       hipMemcpyHostToDevice));

  NIDType* d_root_nodes = reinterpret_cast<NIDType*>(d_input);
  TimestampType* d_root_timestamps = reinterpret_cast<TimestampType*>(
      d_input + root_nodes.size() * sizeof(NIDType));
  uint32_t* d_cumsum_num_nodes = reinterpret_cast<uint32_t*>(
      d_input + root_nodes.size() * sizeof(NIDType) +
      root_timestamps.size() * sizeof(TimestampType));

  delete[] tmp_host_buffer;

  // device output
  uint32_t num_root_nodes = root_nodes.size();
  std::size_t offset1 = num_root_nodes * fanouts_[layer] * sizeof(NIDType);
  std::size_t offset2 =
      offset1 + num_root_nodes * fanouts_[layer] * sizeof(TimestampType);
  std::size_t offset3 =
      offset2 + num_root_nodes * fanouts_[layer] * sizeof(TimestampType);
  std::size_t offset4 =
      offset3 + num_root_nodes * fanouts_[layer] * sizeof(EIDType);
  std::size_t total_output_size = offset4 + num_root_nodes * sizeof(uint32_t);

  char* d_output = reinterpret_cast<char*>(mr->allocate(total_output_size));

  NIDType* d_src_nodes = reinterpret_cast<NIDType*>(d_output);
  TimestampType* d_timestamps =
      reinterpret_cast<TimestampType*>(d_output + offset1);
  TimestampType* d_delta_timestamps =
      reinterpret_cast<TimestampType*>(d_output + offset2);
  EIDType* d_eids = reinterpret_cast<EIDType*>(d_output + offset3);
  uint32_t* d_num_sampled = reinterpret_cast<uint32_t*>(d_output + offset4);

  uint32_t num_threads_per_block = 256;
  uint32_t num_blocks =
      (num_root_nodes + num_threads_per_block - 1) / num_threads_per_block;

  if (sampling_policy_ == SamplingPolicy::kSamplingPolicyRecent) {
    SampleLayerRecentKernel<<<num_blocks, num_threads_per_block>>>(
        graph_.get_device_node_table(), graph_.num_nodes(), prop_time_,
        d_root_nodes, d_root_timestamps, d_cumsum_num_nodes, num_snapshots_,
        snapshot_time_window_, num_root_nodes, fanouts_[layer], d_src_nodes,
        d_timestamps, d_delta_timestamps, d_eids, d_num_sampled);
  } else if (sampling_policy_ == SamplingPolicy::kSamplingPolicyUniform) {
    rmm::device_vector<hiprandState_t> d_rand_states(num_threads_per_block *
                                                    num_blocks);
    auto rand_states = thrust::raw_pointer_cast(d_rand_states.data());

    auto max_shared_memory_size = GetSharedMemoryMaxSize();
    int offset_per_thread =
        max_shared_memory_size / sizeof(SamplingRange) / num_threads_per_block;

    LOG(DEBUG) << "Max shared memory size: " << max_shared_memory_size
               << " bytes"
               << ", offset per thread: " << offset_per_thread;

    // launch sampling kernel
    SampleLayerUniformKernel<<<num_blocks, num_threads_per_block,
                               offset_per_thread * num_threads_per_block *
                                   sizeof(SamplingRange)>>>(
        graph_.get_device_node_table(), graph_.num_nodes(), prop_time_,
        rand_states, seed_, offset_per_thread, d_root_nodes, d_root_timestamps,
        d_cumsum_num_nodes, num_snapshots_, snapshot_time_window_,
        num_root_nodes, fanouts_[layer], d_src_nodes, d_timestamps,
        d_delta_timestamps, d_eids, d_num_sampled);
  }

  // copy output to host
  char* tmp_host_buffer_output = new char[total_output_size];
  CUDA_CALL(hipMemcpy(tmp_host_buffer_output, d_output, total_output_size,
                       hipMemcpyDeviceToHost));

  // host output
  std::vector<NIDType> src_nodes(
      reinterpret_cast<NIDType*>(tmp_host_buffer_output),
      reinterpret_cast<NIDType*>(tmp_host_buffer_output + offset2));
  std::vector<TimestampType> timestamps(
      reinterpret_cast<TimestampType*>(tmp_host_buffer_output + offset1),
      reinterpret_cast<TimestampType*>(tmp_host_buffer_output + offset2));
  std::vector<TimestampType> delta_timestamps(
      reinterpret_cast<TimestampType*>(tmp_host_buffer_output + offset2),
      reinterpret_cast<TimestampType*>(tmp_host_buffer_output + offset3));
  std::vector<EIDType> eids(
      reinterpret_cast<EIDType*>(tmp_host_buffer_output + offset3),
      reinterpret_cast<EIDType*>(tmp_host_buffer_output + offset4));
  std::vector<uint32_t> num_sampled(
      reinterpret_cast<uint32_t*>(tmp_host_buffer_output + offset4),
      reinterpret_cast<uint32_t*>(tmp_host_buffer_output + total_output_size));

  delete[] tmp_host_buffer_output;

  // convert to SamplingResult
  std::vector<SamplingResult> sampling_results(num_snapshots_);
  uint32_t snapshot_offset = 0;
  for (int snapshot = 0; snapshot < num_snapshots_; ++snapshot) {
    auto& sampling_result = sampling_results[snapshot];

    // copy dst nodes
    std::copy(prev_sampling_results.at(snapshot).all_nodes.begin(),
              prev_sampling_results.at(snapshot).all_nodes.end(),
              std::back_inserter(sampling_result.all_nodes));
    std::copy(prev_sampling_results.at(snapshot).all_timestamps.begin(),
              prev_sampling_results.at(snapshot).all_timestamps.end(),
              std::back_inserter(sampling_result.all_timestamps));

    uint32_t num_nodes_this_snapshot;
    if (snapshot == 0) {
      num_nodes_this_snapshot = cumsum_num_nodes[0];
    } else {
      num_nodes_this_snapshot =
          cumsum_num_nodes[snapshot] - cumsum_num_nodes[snapshot - 1];
    }

    uint32_t num_sampled_total = 0;
    for (uint32_t i = 0; i < num_nodes_this_snapshot; i++) {
      std::vector<NIDType> row(num_sampled[snapshot_offset + i]);
      std::fill(row.begin(), row.end(), i);
      std::copy(row.begin(), row.end(),
                std::back_inserter(sampling_result.row));

      std::copy(src_nodes.begin() + (snapshot_offset + i) * fanouts_[layer],
                src_nodes.begin() + num_sampled[i] +
                    (snapshot_offset + i) * fanouts_[layer],
                std::back_inserter(sampling_result.all_nodes));

      std::copy(timestamps.begin() + (snapshot_offset + i) * fanouts_[layer],
                timestamps.begin() + num_sampled[i] +
                    (snapshot_offset + i) * fanouts_[layer],
                std::back_inserter(sampling_result.all_timestamps));

      std::copy(
          delta_timestamps.begin() + (snapshot_offset + i) * fanouts_[layer],
          delta_timestamps.begin() + num_sampled[i] +
              (snapshot_offset + i) * fanouts_[layer],
          std::back_inserter(sampling_result.delta_timestamps));

      std::copy(eids.begin() + (snapshot_offset + i) * fanouts_[layer],
                eids.begin() + num_sampled[i] +
                    (snapshot_offset + i) * fanouts_[layer],
                std::back_inserter(sampling_result.eids));

      num_sampled_total += num_sampled[i];
    }
    sampling_result.col.resize(num_sampled_total);
    std::iota(sampling_result.col.begin(), sampling_result.col.end(),
              num_nodes_this_snapshot);

    sampling_result.num_dst_nodes = num_nodes_this_snapshot;
    sampling_result.num_src_nodes = num_nodes_this_snapshot + num_sampled_total;

    snapshot_offset += num_nodes_this_snapshot;
  }

  mr->deallocate(d_input, total_input_size);
  mr->deallocate(d_output, total_output_size);

  return sampling_results;
}

std::vector<std::vector<SamplingResult>> TemporalSampler::Sample(
    const std::vector<NIDType>& dst_nodes,
    const std::vector<TimestampType>& timestamps) {
  CHECK_EQ(dst_nodes.size(), timestamps.size());
  std::vector<std::vector<SamplingResult>> results;

  for (int layer = 0; layer < num_layers_; ++layer) {
    if (layer == 0) {
      results.push_back(
          SampleLayer(layer, RootInputToSamplingResult(dst_nodes, timestamps)));
    } else {
      results.push_back(SampleLayer(layer, results.back()));
    }
  }
  return results;
}
}  // namespace dgnn
