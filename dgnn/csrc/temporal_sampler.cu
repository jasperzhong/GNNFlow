#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdint>
#include <numeric>
#include <rmm/device_vector.hpp>

#include "common.h"
#include "sampling_kernels.h"
#include "temporal_sampler.h"
#include "utils.h"

namespace dgnn {

TemporalSampler::TemporalSampler(const DynamicGraph& graph,
                                 const std::vector<uint32_t>& fanouts,
                                 SamplingPolicy sampling_policy,
                                 uint32_t num_snapshots,
                                 float snapshot_time_window, bool prop_time,
                                 uint64_t seed)
    : graph_(graph),
      fanouts_(fanouts),
      sampling_policy_(sampling_policy),
      num_snapshots_(num_snapshots),
      snapshot_time_window_(snapshot_time_window),
      prop_time_(prop_time),
      num_layers_(fanouts.size()),
      seed_(seed),
      streams_(nullptr),
      cpu_buffer_(nullptr),
      gpu_input_buffer_(nullptr),
      gpu_output_buffer_(nullptr),
      rand_states_(nullptr),
      initialized_(false) {
  if (num_snapshots_ == 1 && std::fabs(snapshot_time_window_) > 0.0f) {
    LOG(WARNING) << "Snapshot time window must be 0 when num_snapshots = 1. "
                    "Ignore the snapshot time window.";
  }
  shared_memory_size_ = GetSharedMemoryMaxSize();

  streams_ = new hipStream_t[num_snapshots_];
  for (uint32_t i = 0; i < num_snapshots_; ++i) {
    CUDA_CALL(hipStreamCreate(&streams_[i]));
  }

  cpu_buffer_ = new char*[num_snapshots_];
  gpu_input_buffer_ = new char*[num_snapshots_];
  gpu_output_buffer_ = new char*[num_snapshots_];
  rand_states_ = new hiprandState_t*[num_snapshots_];
}

TemporalSampler::~TemporalSampler() {
  for (uint32_t i = 0; i < num_snapshots_; ++i) {
    if (cpu_buffer_[i] != nullptr) {
      hipHostFree(cpu_buffer_[i]);
    }

    if (gpu_input_buffer_[i] != nullptr) {
      hipFree(gpu_input_buffer_[i]);
    }

    if (gpu_output_buffer_[i] != nullptr) {
      hipFree(gpu_output_buffer_[i]);
    }

    if (rand_states_[i] != nullptr) {
      hipFree(rand_states_[i]);
    }
  }

  delete[] cpu_buffer_;
  delete[] gpu_input_buffer_;
  delete[] gpu_output_buffer_;
  delete[] rand_states_;

  for (uint32_t i = 0; i < num_snapshots_; ++i) {
    hipStreamDestroy(streams_[i]);
  }
  delete[] streams_;
}

void TemporalSampler::InitBuffer(std::size_t num_root_nodes) {
  std::size_t maximum_sampled_nodes = num_root_nodes;
  for (int i = 0; i < num_layers_; i++) {
    // including itself
    maximum_sampled_nodes += maximum_sampled_nodes * fanouts_[i];
  }
  LOG(DEBUG) << "Maximum sampled nodes: " << maximum_sampled_nodes;

  constexpr std::size_t per_node_size =
      sizeof(NIDType) + sizeof(TimestampType) + sizeof(TimestampType) +
      sizeof(EIDType) + sizeof(uint32_t);

  for (uint32_t i = 0; i < num_snapshots_; ++i) {
    CUDA_CALL(
        hipHostMalloc(&cpu_buffer_[i], per_node_size * maximum_sampled_nodes));

    CUDA_CALL(hipMalloc(&gpu_input_buffer_[i],
                         per_node_size * maximum_sampled_nodes));
    CUDA_CALL(hipMalloc(&gpu_output_buffer_[i],
                         per_node_size * maximum_sampled_nodes));

    LOG(DEBUG) << "Allocated CPU & GPU buffer: "
               << maximum_sampled_nodes * per_node_size << " bytes";

    if (sampling_policy_ == SamplingPolicy::kSamplingPolicyUniform) {
      CUDA_CALL(hipMalloc((void**)&rand_states_[i],
                           maximum_sampled_nodes * sizeof(hiprandState)));
      uint32_t num_threads_per_block = 256;
      uint32_t num_blocks =
          (maximum_sampled_nodes + num_threads_per_block - 1) /
          num_threads_per_block;

      InitCuRandStates<<<num_blocks, num_threads_per_block>>>(rand_states_[i],
                                                              seed_);
    }
  }

  initialized_ = true;
}

std::vector<SamplingResult> TemporalSampler::RootInputToSamplingResult(
    const std::vector<NIDType>& dst_nodes,
    const std::vector<TimestampType>& dst_timestamps) {
  std::vector<SamplingResult> sampling_results(num_snapshots_);
  for (int snapshot = 0; snapshot < num_snapshots_; ++snapshot) {
    auto& sampling_result = sampling_results[snapshot];
    sampling_result.all_nodes.insert(sampling_result.all_nodes.end(),
                                     dst_nodes.begin(), dst_nodes.end());
    sampling_result.all_timestamps.insert(sampling_result.all_timestamps.end(),
                                          dst_timestamps.begin(),
                                          dst_timestamps.end());
  }
  return sampling_results;
}

std::vector<SamplingResult> TemporalSampler::SampleLayer(
    uint32_t layer, const std::vector<SamplingResult>& prev_sampling_results) {
  CHECK_EQ(prev_sampling_results.size(), num_snapshots_);

  // prepare input
  std::vector<std::size_t> num_root_nodes_list(num_snapshots_);
  for (uint32_t snapshot = 0; snapshot < num_snapshots_; ++snapshot) {
    auto& sampling_result = prev_sampling_results.at(snapshot);
    auto& all_nodes = sampling_result.all_nodes;
    auto& all_timestamps = sampling_result.all_timestamps;
    std::size_t num_root_nodes = all_nodes.size();
    num_root_nodes_list[snapshot] = num_root_nodes;

    char* root_nodes_dst = cpu_buffer_[snapshot];
    char* root_timestamps_dst =
        cpu_buffer_[snapshot] + num_root_nodes * sizeof(NIDType);

    // copy all_nodes and all_timestamps to cpu_buffer_
    Copy(root_nodes_dst, all_nodes.data(), all_nodes.size() * sizeof(NIDType));
    Copy(root_timestamps_dst, all_timestamps.data(),
         all_timestamps.size() * sizeof(TimestampType));

    CUDA_CALL(hipMemcpyAsync(
        gpu_input_buffer_[snapshot], cpu_buffer_[snapshot],
        num_root_nodes * (sizeof(NIDType) + sizeof(TimestampType)),
        hipMemcpyHostToDevice, streams_[snapshot]));
  }

  // launch kernel for each snapshot
  std::vector<std::size_t> total_output_size_list(num_snapshots_);
  for (uint32_t snapshot = 0; snapshot < num_snapshots_; ++snapshot) {
    std::size_t num_root_nodes = num_root_nodes_list[snapshot];

    NIDType* d_root_nodes =
        reinterpret_cast<NIDType*>(gpu_input_buffer_[snapshot]);
    TimestampType* d_root_timestamps = reinterpret_cast<TimestampType*>(
        gpu_input_buffer_[snapshot] + num_root_nodes * sizeof(NIDType));

    // device output
    std::size_t offset1 = num_root_nodes * fanouts_[layer] * sizeof(NIDType);
    std::size_t offset2 =
        offset1 + num_root_nodes * fanouts_[layer] * sizeof(EIDType);
    std::size_t offset3 =
        offset2 + num_root_nodes * fanouts_[layer] * sizeof(TimestampType);
    std::size_t offset4 =
        offset3 + num_root_nodes * fanouts_[layer] * sizeof(TimestampType);
    std::size_t total_output_size = offset4 + num_root_nodes * sizeof(uint32_t);
    total_output_size_list[snapshot] = total_output_size;

    LOG(DEBUG) << "Total output size: " << total_output_size;

    NIDType* d_src_nodes =
        reinterpret_cast<NIDType*>(gpu_output_buffer_[snapshot]);
    EIDType* d_eids =
        reinterpret_cast<EIDType*>(gpu_output_buffer_[snapshot] + offset1);
    TimestampType* d_timestamps = reinterpret_cast<TimestampType*>(
        gpu_output_buffer_[snapshot] + offset2);
    TimestampType* d_delta_timestamps = reinterpret_cast<TimestampType*>(
        gpu_output_buffer_[snapshot] + offset3);
    uint32_t* d_num_sampled =
        reinterpret_cast<uint32_t*>(gpu_output_buffer_[snapshot] + offset4);

    uint32_t num_threads_per_block = 256;
    uint32_t num_blocks =
        (num_root_nodes + num_threads_per_block - 1) / num_threads_per_block;

    if (sampling_policy_ == SamplingPolicy::kSamplingPolicyRecent) {
      SampleLayerRecentKernel<<<num_blocks, num_threads_per_block, 0,
                                streams_[snapshot]>>>(
          graph_.get_device_node_table(), graph_.num_nodes(), prop_time_,
          d_root_nodes, d_root_timestamps, snapshot, num_snapshots_,
          snapshot_time_window_, num_root_nodes, fanouts_[layer], d_src_nodes,
          d_eids, d_timestamps, d_delta_timestamps, d_num_sampled);
    } else if (sampling_policy_ == SamplingPolicy::kSamplingPolicyUniform) {
      int offset_per_thread =
          shared_memory_size_ / sizeof(SamplingRange) / num_threads_per_block;

      LOG(DEBUG) << "Max shared memory size: " << shared_memory_size_
                 << " bytes"
                 << ", offset per thread: " << offset_per_thread;

      SampleLayerUniformKernel<<<num_blocks, num_threads_per_block,
                                 offset_per_thread * num_threads_per_block *
                                     sizeof(SamplingRange),
                                 streams_[snapshot]>>>(
          graph_.get_device_node_table(), graph_.num_nodes(), prop_time_,
          rand_states_[snapshot], seed_, offset_per_thread, d_root_nodes,
          d_root_timestamps, snapshot, num_snapshots_, snapshot_time_window_,
          num_root_nodes, fanouts_[layer], d_src_nodes, d_eids, d_timestamps,
          d_delta_timestamps, d_num_sampled);
    }
  }

  // copy output to host
  for (uint32_t snapshot = 0; snapshot < num_snapshots_; ++snapshot) {
    std::size_t num_root_nodes = num_root_nodes_list[snapshot];
    CUDA_CALL(hipMemcpyAsync(cpu_buffer_[snapshot],
                              gpu_output_buffer_[snapshot],
                              total_output_size_list[snapshot],
                              hipMemcpyDeviceToHost, streams_[snapshot]));
  }

  // combine
  std::vector<SamplingResult> sampling_results(num_snapshots_);
  for (uint32_t snapshot = 0; snapshot < num_snapshots_; ++snapshot) {
    auto& prev_sampling_result = prev_sampling_results.at(snapshot);
    std::size_t num_root_nodes = num_root_nodes_list[snapshot];

    std::size_t offset1 = num_root_nodes * fanouts_[layer] * sizeof(NIDType);
    std::size_t offset2 =
        offset1 + num_root_nodes * fanouts_[layer] * sizeof(EIDType);
    std::size_t offset3 =
        offset2 + num_root_nodes * fanouts_[layer] * sizeof(TimestampType);
    std::size_t offset4 =
        offset3 + num_root_nodes * fanouts_[layer] * sizeof(TimestampType);

    // host output
    NIDType* src_nodes = reinterpret_cast<NIDType*>(cpu_buffer_[snapshot]);
    EIDType* eids = reinterpret_cast<EIDType*>(cpu_buffer_[snapshot] + offset1);
    TimestampType* timestamps =
        reinterpret_cast<TimestampType*>(cpu_buffer_[snapshot] + offset2);
    TimestampType* delta_timestamps =
        reinterpret_cast<TimestampType*>(cpu_buffer_[snapshot] + offset3);
    uint32_t* num_sampled =
        reinterpret_cast<uint32_t*>(cpu_buffer_[snapshot] + offset4);

    auto& sampling_result = sampling_results[snapshot];

    // copy dst nodes
    std::copy(prev_sampling_results.at(snapshot).all_nodes.begin(),
              prev_sampling_results.at(snapshot).all_nodes.end(),
              std::back_inserter(sampling_result.all_nodes));
    std::copy(prev_sampling_results.at(snapshot).all_timestamps.begin(),
              prev_sampling_results.at(snapshot).all_timestamps.end(),
              std::back_inserter(sampling_result.all_timestamps));

    CUDA_CALL(hipStreamSynchronize(streams_[snapshot]));

    uint32_t num_sampled_total = 0;
    for (uint32_t i = 0; i < num_root_nodes; ++i) {
      num_sampled_total += num_sampled[i];
    }

    sampling_result.col.resize(num_sampled_total);
    std::iota(sampling_result.col.begin(), sampling_result.col.end(),
              num_root_nodes);

    sampling_result.num_dst_nodes = num_root_nodes;
    sampling_result.num_src_nodes = num_root_nodes + num_sampled_total;

    sampling_result.all_nodes.reserve(sampling_result.num_src_nodes);
    sampling_result.all_timestamps.reserve(sampling_result.num_src_nodes);
    sampling_result.row.reserve(num_sampled_total);
    sampling_result.delta_timestamps.reserve(num_sampled_total);
    sampling_result.eids.reserve(num_sampled_total);

    for (uint32_t i = 0; i < num_root_nodes; i++) {
      std::vector<NIDType> row(num_sampled[i]);
      std::fill(row.begin(), row.end(), i);
      std::copy(row.begin(), row.end(),
                std::back_inserter(sampling_result.row));

      std::copy(src_nodes + i * fanouts_[layer],
                src_nodes + num_sampled[i] + i * fanouts_[layer],
                std::back_inserter(sampling_result.all_nodes));

      std::copy(timestamps + i * fanouts_[layer],
                timestamps + num_sampled[i] + i * fanouts_[layer],
                std::back_inserter(sampling_result.all_timestamps));

      std::copy(delta_timestamps + i * fanouts_[layer],
                delta_timestamps + num_sampled[i] + i * fanouts_[layer],
                std::back_inserter(sampling_result.delta_timestamps));

      std::copy(eids + i * fanouts_[layer],
                eids + num_sampled[i] + i * fanouts_[layer],
                std::back_inserter(sampling_result.eids));
    }
  }

  return sampling_results;
}

std::vector<std::vector<SamplingResult>> TemporalSampler::Sample(
    const std::vector<NIDType>& dst_nodes,
    const std::vector<TimestampType>& timestamps) {
  CHECK_EQ(dst_nodes.size(), timestamps.size());
  std::vector<std::vector<SamplingResult>> results;

  if (initialized_ == false) InitBuffer(dst_nodes.size());

  for (int layer = 0; layer < num_layers_; ++layer) {
    if (layer == 0) {
      results.push_back(
          SampleLayer(layer, RootInputToSamplingResult(dst_nodes, timestamps)));
    } else {
      results.push_back(SampleLayer(layer, results.back()));
    }
  }
  return results;
}
}  // namespace dgnn
