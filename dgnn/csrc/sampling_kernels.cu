#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include "sampling_kernels.h"

namespace dgnn {

__host__ __device__ void LowerBound(TimestampType* timestamps,
                                    std::size_t num_edges,
                                    TimestampType timestamp, std::size_t* idx) {
  std::size_t left = 0;
  std::size_t right = num_edges;
  while (left < right) {
    std::size_t mid = (left + right) / 2;
    if (timestamps[mid] < timestamp) {
      left = mid + 1;
    } else {
      right = mid;
    }
  }
  *idx = left;
}

__host__ __device__ void UpperBound(TimestampType* timestamps,
                                    std::size_t num_edges,
                                    TimestampType timestamp, std::size_t* idx) {
  std::size_t left = 0;
  std::size_t right = num_edges;
  while (left < right) {
    std::size_t mid = (left + right) / 2;
    if (timestamps[mid] <= timestamp) {
      left = mid + 1;
    } else {
      right = mid;
    }
  }
  *idx = left;
}

__device__ void QuickSort(uint32_t* indices, int lo, int hi) {
  if (lo >= hi || lo < 0 || hi < 0) return;
  int i = lo, j = hi;
  int mid = (lo + hi) / 2;
  uint32_t pivot = indices[mid];
  while (i <= j) {
    while (indices[i] < pivot) i++;
    while (indices[j] > pivot) j--;
    if (i >= j) break;
    uint32_t tmp = indices[i];
    indices[i] = indices[j];
    indices[j] = tmp;
  }
  QuickSort(indices, lo, j);
  QuickSort(indices, j, hi);
}

struct SamplingRangeInBlock {
  // [start_idx, end_idx)
  TemporalBlock* block;
  std::size_t start_idx;
  std::size_t end_idx;

  __host__ __device__ SamplingRangeInBlock()
      : block(nullptr), start_idx(0), end_idx(0) {}
};

__global__ void SampleLayerFromRootKernel(
    const DoublyLinkedList* node_table, std::size_t num_nodes,
    SamplingPolicy sampling_policy, hiprandState_t* rand_states, uint64_t seed,
    NIDType* root_nodes, TimestampType* start_timestamps,
    TimestampType* end_timestamps, std::size_t num_dst_nodes, uint32_t fanout,
    NIDType* src_nodes, TimestampType* timestamps,
    TimestampType* delta_timestamps, EIDType* eids, uint32_t* num_sampled) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_dst_nodes) {
    return;
  }

  NIDType nid = root_nodes[tid];
  TimestampType start_timestamp = start_timestamps[tid];
  TimestampType end_timestamp = end_timestamps[tid];

  auto& list = node_table[nid];
  uint32_t num_candidates = 0;

  SamplingRangeInBlock* sampling_range = new SamplingRangeInBlock[list.size];
  auto curr = list.head;
  uint32_t curr_idx = 0;
  while (curr != nullptr) {
    if (end_timestamp < curr->timestamps[0]) {
      // search in the next block
      curr = curr->next;
      curr_idx++;
      continue;
    }

    if (start_timestamp > curr->timestamps[curr->size - 1]) {
      // no need to search in the next block
      break;
    }

    // search in the current block
    if (start_timestamp >= curr->timestamps[0] &&
        end_timestamp <= curr->timestamps[curr->size - 1]) {
      // all edges in the current block
      LowerBound(curr->timestamps, curr->size, start_timestamp,
                 &sampling_range[curr_idx].start_idx);
      UpperBound(curr->timestamps, curr->size, end_timestamp,
                 &sampling_range[curr_idx].end_idx);
      sampling_range[curr_idx].block = curr;

      num_candidates +=
          sampling_range[curr_idx].end_idx - sampling_range[curr_idx].start_idx;
      break;
    } else if (start_timestamp < curr->timestamps[0] &&
               end_timestamp >= curr->timestamps[curr->size - 1]) {
      // only the edges before end_timestamp are in the current block
      sampling_range[curr_idx].start_idx = 0;
      UpperBound(curr->timestamps, curr->size, end_timestamp,
                 &sampling_range[curr_idx].end_idx);
      sampling_range[curr_idx].block = curr;

      num_candidates +=
          sampling_range[curr_idx].end_idx - sampling_range[curr_idx].start_idx;
      curr = curr->next;
      curr_idx++;
      continue;
    } else if (start_timestamp >= curr->timestamps[0] &&
               end_timestamp > curr->timestamps[curr->size - 1]) {
      // only the edges after start_timestamp are in the current block
      LowerBound(curr->timestamps, curr->size, start_timestamp,
                 &sampling_range[curr_idx].start_idx);
      sampling_range[curr_idx].end_idx = curr->size;
      sampling_range[curr_idx].block = curr;

      num_candidates +=
          sampling_range[curr_idx].end_idx - sampling_range[curr_idx].start_idx;
      break;
    } else {
      // the whole block is in the range
      sampling_range[curr_idx].start_idx = 0;
      sampling_range[curr_idx].end_idx = curr->size;
      sampling_range[curr_idx].block = curr;

      num_candidates += curr->size;
      curr = curr->next;
      curr_idx++;
      continue;
    }
  }

  uint32_t* indices = new uint32_t[fanout];
  if (sampling_policy == SamplingPolicy::kSamplingPolicyRecent) {
    for (uint32_t i = 0; i < fanout; i++) {
      indices[i] = i;
    }
  } else if (sampling_policy == SamplingPolicy::kSamplingPolicyUniform) {
    hiprand_init(seed, tid, 0, &rand_states[tid]);
    for (uint32_t i = 0; i < fanout; i++) {
      indices[i] = hiprand(rand_states + tid) % num_candidates;
    }
    QuickSort(indices, 0, fanout - 1);
  }

  uint32_t cumsum = 0;
  uint32_t j = 0;
  uint32_t offset = tid * fanout;
  for (uint32_t i = 0; i < list.size; i++) {
    if (sampling_range[i].block == nullptr) {
      continue;
    }
    auto idx = indices[j] - cumsum;
    auto start_idx = sampling_range[i].start_idx;
    auto end_idx = sampling_range[i].end_idx;

    while (j < fanout && idx < end_idx - start_idx) {
      // start from end_idx (newer edges)
      src_nodes[offset + j] =
          sampling_range[i].block->dst_nodes[end_idx - idx - 1];
      timestamps[offset + j] =
          sampling_range[i].block->timestamps[end_idx - idx - 1];
      delta_timestamps[offset + j] =
          end_timestamp -
          sampling_range[i].block->timestamps[end_idx - idx - 1];
      eids[offset + j] = sampling_range[i].block->eids[end_idx - idx - 1];
      idx = indices[++j] - cumsum;
    }

    if (j >= fanout) {
      break;
    }
    cumsum += end_idx - start_idx;
  }

  num_sampled[tid] = j;

  delete[] sampling_range;
  delete[] indices;
}

}  // namespace dgnn
