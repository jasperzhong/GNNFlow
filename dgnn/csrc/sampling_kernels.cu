#include "hip/hip_runtime.h"
#include <math.h>

#include "sampling_kernels.h"

namespace dgnn {

__host__ __device__ void LowerBound(TimestampType* timestamps, int num_edges,
                                    TimestampType timestamp, int* idx) {
  int left = 0;
  int right = num_edges;
  while (left < right) {
    int mid = (left + right) / 2;
    if (timestamps[mid] < timestamp) {
      left = mid + 1;
    } else {
      right = mid;
    }
  }
  *idx = left;
}

template <typename T>
__device__ void inline swap(T a, T b) {
  T c(a);
  a = b;
  b = c;
}

__device__ void QuickSort(uint32_t* indices, int lo, int hi) {
  if (lo >= hi || lo < 0 || hi < 0) return;

  uint32_t pivot = indices[hi];
  int i = lo - 1;
  for (int j = lo; j < hi; ++j) {
    if (indices[j] < pivot) {
      swap(indices[++i], indices[j]);
    }
  }
  swap(indices[++i], indices[hi]);

  QuickSort(indices, lo, i - 1);
  QuickSort(indices, i + 1, hi);
}

__global__ void SampleLayerRecentKernel(
    const DoublyLinkedList* node_table, std::size_t num_nodes, bool prop_time,
    const NIDType* root_nodes, const TimestampType* root_timestamps,
    uint32_t snapshot_idx, uint32_t num_snapshots,
    TimestampType snapshot_time_window, uint32_t num_root_nodes,
    uint32_t fanout, NIDType* src_nodes, EIDType* eids,
    TimestampType* timestamps, TimestampType* delta_timestamps,
    uint32_t* num_sampled) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_root_nodes) {
    return;
  }

  NIDType nid = root_nodes[tid];
  TimestampType root_timestamp = root_timestamps[tid];
  TimestampType start_timestamp, end_timestamp;
  if (num_snapshots == 1) {
    start_timestamp = 0;
    end_timestamp = root_timestamp;
  } else {
    end_timestamp = root_timestamp -
                    (num_snapshots - snapshot_idx - 1) * snapshot_time_window;
    start_timestamp = end_timestamp - snapshot_time_window;
  }

  auto curr = node_table[nid].head;
  uint32_t offset = tid * fanout;
  int start_idx, end_idx;
  uint32_t sampled = 0;
  while (curr != nullptr && sampled < fanout) {
    if (end_timestamp < curr->timestamps[0]) {
      // search in the next block
      curr = curr->next;
      continue;
    }

    if (start_timestamp > curr->timestamps[curr->size - 1]) {
      // no need to search in the next block
      break;
    }

    // search in the current block
    if (start_timestamp >= curr->timestamps[0] &&
        end_timestamp <= curr->timestamps[curr->size - 1]) {
      // all edges in the current block
      LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
      LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
    } else if (start_timestamp < curr->timestamps[0] &&
               end_timestamp < curr->timestamps[curr->size - 1]) {
      // only the edges before end_timestamp are in the current block
      start_idx = 0;
      LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
    } else if (start_timestamp > curr->timestamps[0] &&
               end_timestamp > curr->timestamps[curr->size - 1]) {
      // only the edges after start_timestamp are in the current block
      LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
      end_idx = curr->size;
    } else {
      // the whole block is in the range
      start_idx = 0;
      end_idx = curr->size;
    }

    // copy the edges to the output
    for (int i = end_idx - 1; sampled < fanout && i >= start_idx; --i) {
      src_nodes[offset + sampled] = curr->dst_nodes[i];
      eids[offset + sampled] = curr->eids[i];
      timestamps[offset + sampled] =
          prop_time ? root_timestamp : curr->timestamps[i];
      delta_timestamps[offset + sampled] = root_timestamp - curr->timestamps[i];
      ++sampled;
    }

    curr = curr->next;
  }

  num_sampled[tid] = sampled;

  while (sampled < fanout) {
    src_nodes[offset + sampled] = kInvalidNID;
    ++sampled;
  }
}

__global__ void SampleLayerUniformKernel(
    const DoublyLinkedList* node_table, std::size_t num_nodes, bool prop_time,
    hiprandState_t* rand_states, uint64_t seed, uint32_t offset_per_thread,
    const NIDType* root_nodes, const TimestampType* root_timestamps,
    uint32_t snapshot_idx, uint32_t num_snapshots,
    TimestampType snapshot_time_window, uint32_t num_root_nodes,
    uint32_t fanout, NIDType* src_nodes, EIDType* eids,
    TimestampType* timestamps, TimestampType* delta_timestamps,
    uint32_t* num_sampled) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_root_nodes) {
    return;
  }

  extern __shared__ SamplingRange ranges[];

  NIDType nid = root_nodes[tid];
  TimestampType root_timestamp = root_timestamps[tid];
  TimestampType start_timestamp, end_timestamp;
  if (num_snapshots == 1) {
    start_timestamp = 0;
    end_timestamp = root_timestamp;
  } else {
    end_timestamp = root_timestamp -
                    (num_snapshots - snapshot_idx - 1) * snapshot_time_window;
    start_timestamp = end_timestamp - snapshot_time_window;
  }

  auto& list = node_table[nid];
  uint32_t num_candidates = 0;

  auto curr = list.head;
  int start_idx, end_idx;
  int curr_idx = 0;
  int offset_by_thread = offset_per_thread * threadIdx.x;
  while (curr != nullptr) {
    if (end_timestamp < curr->timestamps[0]) {
      // search in the next block
      curr = curr->next;
      curr_idx += 1;
      continue;
    }

    if (start_timestamp > curr->timestamps[curr->size - 1]) {
      // no need to search in the next block
      break;
    }

    // search in the current block
    if (start_timestamp >= curr->timestamps[0] &&
        end_timestamp <= curr->timestamps[curr->size - 1]) {
      // all edges in the current block
      LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
      LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
    } else if (start_timestamp < curr->timestamps[0] &&
               end_timestamp < curr->timestamps[curr->size - 1]) {
      // only the edges before end_timestamp are in the current block
      start_idx = 0;
      LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
    } else if (start_timestamp > curr->timestamps[0] &&
               end_timestamp > curr->timestamps[curr->size - 1]) {
      // only the edges after start_timestamp are in the current block
      LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
      end_idx = curr->size;
    } else {
      // the whole block is in the range
      start_idx = 0;
      end_idx = curr->size;
    }

    if (curr_idx < offset_per_thread) {
      ranges[offset_by_thread + curr_idx].start_idx = start_idx;
      ranges[offset_by_thread + curr_idx].end_idx = end_idx;
    }

    num_candidates += end_idx - start_idx;
    curr = curr->next;
    curr_idx += 1;
  }

  uint32_t indices[MAX_FANOUT];
  uint32_t to_sample = min(fanout, num_candidates);
  for (uint32_t i = 0; i < to_sample; i++) {
    indices[i] = hiprand(rand_states + tid) % num_candidates;
  }
  QuickSort(indices, 0, to_sample - 1);

  uint32_t sampled = 0;
  uint32_t offset = tid * fanout;

  curr = list.head;
  curr_idx = 0;
  uint32_t cumsum = 0;
  while (curr != nullptr) {
    if (end_timestamp < curr->timestamps[0]) {
      // search in the next block
      curr = curr->next;
      curr_idx += 1;
      continue;
    }

    if (start_timestamp > curr->timestamps[curr->size - 1]) {
      // no need to search in the next block
      break;
    }

    if (curr_idx < offset_per_thread) {
      start_idx = ranges[offset_by_thread + curr_idx].start_idx;
      end_idx = ranges[offset_by_thread + curr_idx].end_idx;
    } else {
      // search in the current block
      if (start_timestamp >= curr->timestamps[0] &&
          end_timestamp <= curr->timestamps[curr->size - 1]) {
        // all edges in the current block
        LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
        LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
      } else if (start_timestamp < curr->timestamps[0] &&
                 end_timestamp < curr->timestamps[curr->size - 1]) {
        // only the edges before end_timestamp are in the current block
        start_idx = 0;
        LowerBound(curr->timestamps, curr->size, end_timestamp, &end_idx);
      } else if (start_timestamp > curr->timestamps[0] &&
                 end_timestamp > curr->timestamps[curr->size - 1]) {
        // only the edges after start_timestamp are in the current block
        LowerBound(curr->timestamps, curr->size, start_timestamp, &start_idx);
        end_idx = curr->size;
      } else {
        // the whole block is in the range
        start_idx = 0;
        end_idx = curr->size;
      }
    }

    auto idx = indices[sampled] - cumsum;
    while (sampled < to_sample && idx < end_idx - start_idx) {
      // start from end_idx (newer edges)
      src_nodes[offset + sampled] = curr->dst_nodes[end_idx - idx - 1];
      eids[offset + sampled] = curr->eids[end_idx - idx - 1];
      timestamps[offset + sampled] =
          prop_time ? root_timestamp : curr->timestamps[end_idx - idx - 1];
      delta_timestamps[offset + sampled] =
          root_timestamp - curr->timestamps[end_idx - idx - 1];
      idx = indices[sampled] - cumsum;
      ++sampled;
    }

    if (sampled >= to_sample) {
      break;
    }

    cumsum += end_idx - start_idx;
    curr = curr->next;
    curr_idx += 1;
  }

  num_sampled[tid] = sampled;

  while (sampled < fanout) {
    src_nodes[offset + sampled] = kInvalidNID;
    ++sampled;
  }
}

}  // namespace dgnn
