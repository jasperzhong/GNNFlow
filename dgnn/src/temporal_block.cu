#include "hip/hip_runtime.h"
#include <thrust/device_delete.h>

#include "temporal_block.h"

namespace dgnn {

__host__ __device__ void InsertBlockToDoublyLinkedList(
    DoublyLinkedList* node_table, NIDType node_id, TemporalBlock* block) {
  auto& list = node_table[node_id];
  auto head_next = list.head.next;
  list.head.next = block;
  block->prev = &list.head;
  block->next = head_next;
  head_next->prev = block;
  list.size++;
}

__global__ void InsertBlockToDoublyLinkedListKernel(
    DoublyLinkedList* node_table, NIDType node_id, TemporalBlock* block) {
  InsertBlockToDoublyLinkedList(node_table, node_id, block);
}

__host__ __device__ void ReplaceBlockInDoublyLinkedList(
    DoublyLinkedList* node_table, NIDType node_id, TemporalBlock* block) {
  auto& list = node_table[node_id];
  auto to_delete = list.head.next;
  list.head.next = block;
  block->prev = &list.head;
  block->next = to_delete->next;
  to_delete->next->prev = block;
}

__global__ void ReplaceBlockInDoublyLinkedListKernel(
    DoublyLinkedList* node_table, NIDType node_id, TemporalBlock* block) {
  ReplaceBlockInDoublyLinkedList(node_table, node_id, block);
}

__host__ __device__ void DeleteTailFromDoublyLinkedList(
    DoublyLinkedList* node_table, NIDType node_id) {
  auto& list = node_table[node_id];
  auto tail = list.tail.prev;
  tail->prev->next = &list.tail;
  list.tail.prev = tail->prev;
  list.size--;
}

__global__ void DeleteTailFromDoublyLinkedListKernel(
    DoublyLinkedList* node_table, NIDType node_id) {
  DeleteTailFromDoublyLinkedList(node_table, node_id);
}

}  // namespace dgnn
